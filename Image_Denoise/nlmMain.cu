
// Host call
void nlmDenoise(const float* h_image_in, float* h_image_out, int width, int height) {

    float* d_image_in;  //image arrays
    float* d_image_out;
    size_t imageSize = width * height * sizeof(float);

    // Allocate gpu mem for calc
    hipMalloc(&d_image_in, imageSize);
    hipMalloc(&d_image_out, imageSize);

    // Copy image to device
    hipMemcpy(d_image_in, h_image_in, imageSize, hipMemcpyHostToDevice);

    // Set up timing events for perfomance bench
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Define grid and block size
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Start timing
    hipEventRecord(start);

    // Make kernel call
    nlmDenoise<<<gridSize, blockSize>>>(d_image_in, d_image_out, width, height);

    // Stop timing
    float ms = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ms, start, stop);
    printf("Execution time: %f ms\n", ms);

    // Copy image back to computer
    hipMemcpy(h_image_out, d_image_out, imageSize, hipMemcpyDeviceToHost);

    // Free mem
    hipFree(d_image_in);
    hipFree(d_image_out);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

