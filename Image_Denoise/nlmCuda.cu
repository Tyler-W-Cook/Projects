#include "hip/hip_runtime.h"
#include <stdio.h>

#define BLOCK_SIZE 16  

#define FILTER_RADIUS 2 
#define SEARCH_RADIUS 5 
#define STANDARD_DEV 10.0    

// CUDA kernel for non-local means denoising
__global__ void nlmDenoise(const float* input, float* output, int width, int height) {

    //Indexing of threads
    int x = blockIj.x * blockDim.x + threadIj.x;
    int y = blockIj.y * blockDim.y + threadIj.y;

    //boundary check
    if (x >= width || y >= height) return;

    float filteredSum = 0.0;                //filtered value Sum
    float normSum = 0.0;                    //normalized Sum 

    float noisy_value = input[y * width + x];       //pixel value to be denoised

    for (int i = 0-x; i <= height; i++) {             // i is searhcing in the y
        for (int j = 0-y; j <= width; j++) {         // j is searching in the x
            int nx = x + j;
            int ny = y + i;

            // Boundary check to ensure the values are valid
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                float otherPixel = input[ny * width + nx];
                float weight = expf(-(noisy_value - otherPixel) * (noisy_value - otherPixel) / (STANDARD_DEV * STANDARD_DEV));
                filteredSum += weight * otherPixel;
                normSum += weight;
            }
        }
    }

    output[y * width + x] = filteredSum / normSum;
}
